#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdio>
#include <iostream>
#include <fstream>
#include <iomanip>
#include "nlohmann/json.hpp"
#include ""

using namespace std;
using json = nlohmann::json;

const int MAX_STRING_LENGTH = 256;
const int THREADS = 3;
#define DATA_FILE "IFF-7_2_MilisiunasJ_L1_dat_1.json"
#define RESULTS_FILE "a.json"

struct Car {
    char brand[MAX_STRING_LENGTH];
    int make_year;
    double mileage;

    void from_json(json data) {
        string brand_temp = data["brand"];
        memset(brand, 0, sizeof(brand));
        strcpy(brand, brand_temp.c_str());

        make_year = data["makeYear"];
        mileage = data["mileage"];
    }

    void print() {
        printf("Brand: %s Make Year: %d Mileage: %f\n", brand, make_year, mileage);
    }
};

void readCarsFile(vector<Car>* cars);
void write_results_to_file(Car* cars, int n, const string file_path, const string title);

__global__ void sum_on_gpu(Car* cars, int* n, int* chunk_size, Car* results);
__device__ void gpu_memset(char* dest);
__device__ void gpu_strcat(char* dest, char* src, int filter);
__device__ char gpu_filter(int src);
char* gpu_convertIntegerToChar(int N);
int main() {
    vector<Car> all_cars;
    readCarsFile(&all_cars);


    printf("%x", gpu_convertIntegerToChar(2015));

    // Host
    Car* cars = &all_cars[0];
    Car results[THREADS];
    int n = all_cars.size();
    int chunk_size = n / THREADS;

    // GPU
    Car* d_all_cars;
    int* d_n;
    int* d_chunk_size;
    Car* d_results;

    // Memory allocation for GPU
    hipMalloc((void**)&d_all_cars, n * sizeof(Car));
    hipMalloc((void**)&d_results, THREADS * sizeof(Car));
    hipMalloc((void**)&d_n, sizeof(int));
    hipMalloc((void**)&d_chunk_size, sizeof(int));

    // Copies memory from CPU to GPU
    hipMemcpy(d_all_cars, cars, n * sizeof(Car), hipMemcpyHostToDevice);
    hipMemcpy(d_n, &n, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_chunk_size, &chunk_size, sizeof(int), hipMemcpyHostToDevice);

    sum_on_gpu <<< 1, THREADS >>> (d_all_cars, d_n, d_chunk_size, d_results);
    printf("\nKiekvienos gijos susumuoti duomenys:\n");
    hipDeviceSynchronize();

    hipMemcpy(&results, d_results, THREADS * sizeof(Car), hipMemcpyDeviceToHost);
    hipFree(d_all_cars);
    hipFree(d_n);
    hipFree(d_chunk_size);
    hipFree(d_results);

    cout << "Finished" << endl;
    write_results_to_file(results, THREADS, RESULTS_FILE, "A dalies rezultatai");
    return 0;
}

/**
 * GPU
 * Sums cars list chunk data properties
 * @param cars Cars list
 * @param n Cars list size
 * @param chunk_size Summed items per thread
 * @param results Summed chunk results
 */
__global__ void sum_on_gpu(Car* cars, int* n, int* chunk_size, Car* results) {
    int start_index = threadIdx.x * *chunk_size;
    int end_index = min(start_index + *chunk_size, *n);

    if (end_index + *chunk_size >= *n || threadIdx.x + 1 == THREADS) {
        end_index = *n;
    }

    Car sum;
    gpu_memset(sum.brand);
    sum.make_year = 0;
    sum.mileage = 0.0;

    for (int i = start_index; i < end_index; ++i) {
        if (i < *n) {
            gpu_strcat(sum.brand, cars[i].brand, cars[i].make_year);
            sum.make_year += cars[i].make_year;
            sum.mileage += cars[i].mileage;
        }
    }

    //    printf("Thread: %d Start Index: %d End Index: %d\n", threadIdx.x, start_index, end_index);
    results[threadIdx.x] = sum;
    printf("Thread: %d Brand: %s Make Year: %d Mileage: %f\n", threadIdx.x, results[threadIdx.x].brand, results[threadIdx.x].make_year, results[threadIdx.x].mileage);
}

/**
 * Appends char array to other char array
 * @param dest Destination array
 * @param src Source array
 */
__device__ void gpu_strcat(char* dest, char* src, int filter) {
    for (int i = 0; i < MAX_STRING_LENGTH; ++i) {
        if (dest[i] == 0) {
            for (int j = 0; j < MAX_STRING_LENGTH; ++j) {
                if (src[j] != 0) {
                    dest[i + j] = src[j];
                }
                else {
                    dest[i + j] = '-';
                    dest[i + j + 1] = gpu_filter(filter);
                    dest[i + j + 2] = ' ';
                    dest[i + j + 3] = ' ';
                    dest[i + j + 4] = ' ';
                    break;
                }
            }
            break;
        }
    }
}


__device__ char gpu_filter(int src) {

    char filter;
    if (src > 2015) {
        filter = 'A';
    }
    else {
        filter = 'B';
    }
    return filter;
}




/**
 * Zeroes all char memory
 * @param dest Char array
 */
__device__ void gpu_memset(char* dest) {
    for (int i = 0; i < MAX_STRING_LENGTH; ++i) {
        dest[i] = 0;
    }
}




// Function to convert integer to
// character array
char* gpu_convertIntegerToChar(int N)
{

    char result[sizeof(N)];


    int tmp;
    int p = 0;
    tmp = N;
        while (tmp != 0)
        {
            result[p] = (tmp % 10);
            tmp = (tmp - result[p]) / 10;
            p++;
        }
        return result;

}






/**
 * Reads cards data in isjon format from the file,
 * parses to Car structs
 * @param cars Destination list
 */
void readCarsFile(vector<Car>* cars) {
    ifstream stream(DATA_FILE);
    json allCarsJson = json::parse(stream);
    auto allCars = allCarsJson["cars"];
    printf("Pradiniai duomenys:\n");
    for (const json& new_car : allCars) {
        Car tempCar;
        tempCar.from_json(new_car);
        tempCar.print();
        cars->push_back(tempCar);
    }
}

/**
 * Writes given monitor cars formatted in table to file
 * @param cars Cars list
 * @param file_path Result file path
 * @param title Results table title
 */
void write_results_to_file(Car* cars, int n, const string file_path, const string title) {
    ofstream file;
    file.open(file_path);
    file << setw(70) << title << endl
        << "------------------------------------------------------------------------------------------------------------------------"
        << endl
        << setw(60) << "Gamintojas |" << setw(30) << "Pagaminimo metai |" << setw(30) << "Rida" << endl
        << "------------------------------------------------------------------------------------------------------------------------"
        << endl;
    for (int i = 0; i < n; ++i) {
        string str(cars[i].brand);
        file << setw(60) << str + " |" << setw(30) << to_string(cars[i].make_year) + " |"
            << setw(30)
            << to_string(cars[i].mileage) << endl;
    }

    file << endl << endl << endl;
}